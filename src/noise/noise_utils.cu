#include "hip/hip_runtime.h"
#include "../../include/noise/noise_utils.h"
#include "../../include/noise/perlin_noise.h"
#include <math.h>

// Enhanced noise function with better distribution of values
__device__ float enhancedNoise(float x, float y, float z) {
    float val = noise(x, y, z);
    val = (val + 1.0f) * 0.5f;
    // Use a milder transformation
    val = powf(val, 0.9f);
    return val;
}

__device__ float distributedNoise(float x, float y, float z, int octaves) {
    float total = 0.0f;
    float frequency = 1.0f;
    float amplitude = 1.0f;
    float maxValue = 0.0f;
    
    for(int i = 0; i < octaves; i++) {
        total += enhancedNoise(x * frequency, y * frequency, z) * amplitude;
        maxValue += amplitude;
        amplitude *= 0.6f;  // slower decay
        frequency *= 1.9f;  // prime-based multiplier instead of 2.0f
    }
    
    total /= maxValue;
    return total;
}