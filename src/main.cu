#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../include/terrain_types.h"
#include "../include/perlin_noise.h"
#include "../include/terrain_gen.h"
#include "../include/visualization.h"

int main() {
    // Initialize terrain types
    TerrainTypes::initializeTerrainTypes();

    // Define terrain size
    int width = 1024;
    int height = 1024;
    int size = width * height * sizeof(int);
    int imageSize = width * height * 3 * sizeof(unsigned char); // RGB
    
    // Allocate host memory
    int* h_terrain = (int*)malloc(size);
    unsigned char* h_image = (unsigned char*)malloc(imageSize);
    
    // Allocate device memory
    int* d_terrain;
    unsigned char* d_image;
    hipMalloc(&d_terrain, size);
    hipMalloc(&d_image, imageSize);
    
    // Generate terrain
    createPerlinNoiseTerrain(d_terrain, width, height);
    
    // Visualize terrain
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    visualizeTerrain<<<gridSize, blockSize>>>(d_terrain, d_image, width, height);
    
    // Copy results back to host
    hipMemcpy(h_terrain, d_terrain, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);
    
    // Save image to file
    saveToPPM("terrain.ppm", h_image, width, height);
    
    // Clean up
    free(h_terrain);
    free(h_image);
    hipFree(d_terrain);
    hipFree(d_image);
    
    return 0;
}