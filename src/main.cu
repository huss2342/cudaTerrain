#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../include/terrain_types.h"
#include "../include/perlin_noise.h"
#include "../include/terrain_gen.h"
#include "../include/visualization.h"

int main() {
    // generate a random seed then split it to X and Y offsets
    

    int seed = time(NULL);
    srand(seed);
    // seed = 123; // Uncomment for reproducibility

    float randomOffsetX = (seed % 100) * 1.27f;
    float randomOffsetY = (seed % 100) * 2.53f; 

    printf("Generated terrain with seed: %d\n", seed);
    printf("Offsets: X=%f, Y=%f\n", randomOffsetX, randomOffsetY);

    // Initialize terrain types
    TerrainTypes::initializeTerrainTypes();

    // Define terrain size
    int width = 1024;
    int height = 1024;
    int size = width * height * sizeof(int);
    int imageSize = width * height * 3 * sizeof(unsigned char); // RGB
    
    // Allocate host memory
    int* h_terrain = (int*)malloc(size);
    unsigned char* h_image = (unsigned char*)malloc(imageSize);
    
    // Allocate device memory
    int* d_terrain;
    unsigned char* d_image;
    hipMalloc(&d_terrain, size);
    hipMalloc(&d_image, imageSize);
    int scale = 0.1f;
    printf("Enter scale for terrain generation (e.g., 0.1): ");
    std::cin >> scale; 

    // Generate terrain
    createPerlinNoiseTerrain(d_terrain, width, height, scale, randomOffsetX, randomOffsetY);
    std::cout << "Terrain generated with scale: " << scale << std::endl;

    // Visualize terrain
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    visualizeTerrain<<<gridSize, blockSize>>>(d_terrain, d_image, width, height);
    std::cout << "Terrain visualization kernel launched." << std::endl;

    // Copy results back to host
    hipMemcpy(h_terrain, d_terrain, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);
    
    // Save image to file
    saveToPPM("terrain.ppm", h_image, width, height);
    
    // Clean up
    free(h_terrain);
    free(h_image);
    hipFree(d_terrain);
    hipFree(d_image);
    
    return 0;
}