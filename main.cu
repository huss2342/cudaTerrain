#include "hip/hip_runtime.h"
#include <stdio.h>

// CUDA kernel function to print "Hello World"
// __global__ void helloFromGPU()
// {
//     // Get the thread ID
//     int threadId = threadIdx.x;
//     printf("Hello World from GPU thread %d!\n", threadId);
// }

struct TerrainType {
    int id;
    const char* name;
    Color color;
};

// terrain types
namespace TerrainTypes {
    __constant__ TerrainType WATER    = {0, "Water", {0, 0, 255}};
    __constant__ TerrainType SAND     = {1, "Sand", {255, 255, 0}};
    __constant__ TerrainType GRASS    = {2, "Grass", {0, 255, 0}};
    __constant__ TerrainType ROCK     = {3, "Rock", {128, 128, 128}};
    __constant__ TerrainType SNOW     = {4, "Snow", {255, 255, 255}};
    __constant__ TerrainType LAVA     = {5, "Lava", {255, 0, 0}};
    __constant__ TerrainType ICE      = {6, "Ice", {0, 255, 255}};
    __constant__ TerrainType MUD      = {7, "Mud", {139, 69, 19}};
    __constant__ TerrainType FOREST   = {8, "Forest", {0, 100, 0}};
    __constant__ TerrainType DESERT   = {9, "Desert", {255, 165, 0}};
    __constant__ TerrainType MOUNTAIN = {10, "Mountain", {139, 69, 19}};
    __constant__ TerrainType SWAMP    = {11, "Swamp", {46, 139, 87}};
    __constant__ TerrainType JUNGLE   = {12, "Jungle", {34, 139, 34}};
    __constant__ TerrainType TUNDRA   = {13, "Tundra", {0, 128, 128}};
    __constant__ TerrainType SAVANNA  = {14, "Savanna", {218, 165, 32}};
    __constant__ TerrainType TAIGA    = {15, "Taiga", {0, 128, 0}};
    __constant__ TerrainType STEPPE   = {16, "Steppe", {210, 180, 140}};
    __constant__ TerrainType PRAIRIE  = {17, "Prairie", {255, 228, 196}};
    __constant__ TerrainType PLATEAU  = {18, "Plateau", {205, 133, 63}};
    __constant__ TerrainType CANYON   = {19, "Canyon", {139, 69, 19}};
    __constant__ TerrainType BADLANDS = {20, "Badlands", {139, 69, 19}};
    __constant__ TerrainType MESA     = {21, "Mesa", {139, 69, 19}};
    __constant__ TerrainType OASIS    = {22, "Oasis", {0, 255, 127}};
    __constant__ TerrainType VOLCANO  = {23, "Volcano", {255, 0, 0}};
    __constant__ TerrainType GLACIER  = {24, "Glacier", {0, 255, 255}};
    __constant__ TerrainType FJORD    = {25, "Fjord", {0, 0, 255}};
    __constant__ TerrainType BAY      = {26, "Bay", {0, 0, 255}};
    __constant__ TerrainType COVE     = {27, "Cove", {0, 0, 255}};
    __constant__ TerrainType BEACH    = {28, "Beach", {255, 255, 0}};
    __constant__ TerrainType CLIFF    = {29, "Cliff", {139, 69, 19}};
    __constant__ TerrainType DUNE     = {30, "Dune", {255, 255, 0}};
}


int main() {
    // Print from CPU
    printf("Hello World from CPU!\n");
    
    // Launch the kernel with 5 threads
    helloFromGPU<<<1, 5>>>();
    
    // Wait for GPU to finish before exiting
    hipDeviceSynchronize();
    
    return 0;
}

void renderTerrain(int* terrain, int width, int height) {
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int terrainType = terrain[y * width + x];
            Color color = getColorForTerrainType(terrainType);
            drawPixel(x, y, color);
        }
    }
}